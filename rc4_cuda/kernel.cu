#include "hip/hip_runtime.h"
#include "rc4.h"

/************************************************************************/
/*
The original idea is to obtain one key at a time, decrypt the corresponding ciphertext, and see if the resulting plaintext satisfies a certain condition.
But the process requires too many intermediate variables, and on second thought, the plaintext and ciphertext are heterogeneous or related, so the known plaintext
If the text and the ciphertext are dissimilar, we can get the value of some position of the key stream. This saves a lot of space~~
*/
/************************************************************************/

__device__ void genKey(unsigned char *keyStartAddr, unsigned long long myKeyNum, int *key_len)
{
  /*	char p = maxKeyLen - 1;
    while (myKeyNum && p >=0) {
      keyStartAddr[p--] = (myKeyNum - 1) % keyNum + start;
      myKeyNum = (myKeyNum - 1) / keyNum;
    }
    *key_len = (maxKeyLen - p - 1);
    return keyStartAddr + p + 1;
  */
#if 0
  size_t i = 0;
  while (myKeyNum && i < maxKeyLen)
  {
    keyStartAddr[i++] = (myKeyNum - 1) % keyNum + start;
    myKeyNum = (myKeyNum - 1) / keyNum;
  }
  *key_len = (i);
#endif
#if 1
  *key_len = 5; // XXX
  for (int i = 0; i < *key_len; i++) {
      keyStartAddr[i] = myKeyNum % 256;
      myKeyNum >>= 8;
  }
#endif
}

__global__ void crackRc4Kernel(unsigned char *key, volatile size_t *found)
{
  int keyLen = 0;
  const unsigned long long totalThreadNum = gridDim.x * blockDim.x;
  const unsigned long long keyNum_per_thread = maxNum / totalThreadNum;
  unsigned long long myKeyNum = (threadIdx.x + blockIdx.x * blockDim.x);
  //printf("totalThreadNum: %i\n", totalThreadNum);
  //printf("keyNum_per_thread: %i\n", keyNum_per_thread);
  //printf("myKeyNum: %i\n", myKeyNum);
  bool justIt;
  for (unsigned long long i = 0; i <= keyNum_per_thread; myKeyNum += totalThreadNum, ++i)
  {
    // vKey is a pointer to share_memory
    unsigned char *vKey = (shared_mem + memory_per_thread * threadIdx.x + STATE_LEN);
    genKey(vKey, myKeyNum, &keyLen);

    // Add the salt if it was specified.
    if (saltLen_device > 0)
    {
      memcpy(vKey + keyLen, salt_device, saltLen_device);
      keyLen = keyLen + saltLen_device;
    }

    justIt = device_isKeyRight(vKey, keyLen);

    // Exit if one of the other blocks found it
    if (*found != 0)
      asm("exit;");

    // the current key is not the requested one
    if (justIt)
    {
      // Find the matching key, write it to Host, save the data, modify found, and exit the program
      *found = keyLen;
      memcpy(key, vKey, keyLen);
      key[keyLen] = 0;
      __threadfence();
      asm("exit;");
      break;
    }
  }
}

void cleanup(unsigned char *key_dev, size_t *found_dev)
{
  hipFree(key_dev);
  hipFree(found_dev);
  return;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t crackRc4WithCuda(unsigned char *knownKeyStream_host, size_t knownStreamLen_host, unsigned char *key, size_t *foundLen, unsigned char *saltHost, size_t saltLenHost)
{
  hipError_t cudaStatus;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    return cudaStatus;
  }

  unsigned char *key_dev;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  cudaStatus = hipMalloc((void **)&key_dev, (MAX_KEY_LENGTH + 1) * sizeof(unsigned char));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed!");
    hipFree(key_dev);
    return cudaStatus;
  }

  size_t *found_dev;

  cudaStatus = hipMalloc((void **)&found_dev, sizeof(size_t));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  // Check if the key variable is found
  cudaStatus = hipMemcpy(found_dev, foundLen, sizeof(size_t), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  // Copy constant memory
  cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(salt_device), saltHost, sizeof(unsigned char) * saltLenHost);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpyToSymbol salt failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  cudaStatus = hipMemcpyToSymbol((const void *)&saltLen_device, (const void *)&saltLenHost, sizeof(unsigned char));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpyToSymbol saltLen failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(knowStream_device), knownKeyStream_host, sizeof(unsigned char) * knownStreamLen_host);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpyToSymbol stream failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  cudaStatus = hipMemcpyToSymbol((const void *)&knownStreamLen_device, (const void *)&knownStreamLen_host, sizeof(unsigned char));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpyToSymbol streamlen failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  // Launch a kernel on the GPU with one thread for each element.
  size_t threadNum = (prop.sharedMemPerBlock / MEMORY_PER_THREAD), share_memory = prop.sharedMemPerBlock; // FIXME double check that this works
  if (threadNum > MAX_THREAD_NUM)
  {
    threadNum = MAX_THREAD_NUM;
    share_memory = threadNum * MEMORY_PER_THREAD;
  }

  crackRc4Kernel<<<BLOCK_NUM, threadNum, share_memory>>>(key_dev, found_dev);

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  // Copy output vector from GPU buffer to host memory.
  cudaStatus = hipMemcpy(key, key_dev, (MAX_KEY_LENGTH + 1) * sizeof(unsigned char), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  // Copy output vector from GPU buffer to host memory.
  cudaStatus = hipMemcpy(foundLen, found_dev, sizeof(size_t), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed!");
    cleanup(key_dev, found_dev);
    return cudaStatus;
  }

  return cudaStatus;
}

int main(int argc, char *argv[])
{
  char *fileName = 0, *knownPlainText = 0, *knownKey = 0, *plainText = 0, *noNullSalt = 0, *saltFile = 0, *knownFile = 0;
  unsigned char *actualSalt = 0, *cipherText = 0, *actualPlainText = 0;
  unsigned char *s_box = (unsigned char *)malloc(sizeof(unsigned char) * 256);

  size_t cipherLength = 0, saltLength = 0, keyLength = 0, knownLength = 0;
  // handle options.
  for (size_t myarg = 1; myarg < argc; ++myarg) //(opt = getopt(argc, argv, "F:t:k:p:s:S:l:L:T:I:") != -1))
  {
    fprintf(stderr, "%s\n", argv[myarg]);
    if (argv[myarg][0] != '-' || strlen(argv[myarg]) != 2)
    {
      fprintf(stderr, "Unrecognized option: %s", argv[myarg]);
      return 1;
    }

    int opt = (int)argv[myarg][1];
    switch (opt)
    {
    case 'F':
      fileName = argv[++myarg];
      break;
    case 't':
      knownPlainText = argv[++myarg];
      break;
    case 'k':
      knownKey = argv[++myarg];
      break;
    case 'p':
      plainText = argv[++myarg];
      break;
    case 's':
      noNullSalt = argv[++myarg];
      break;
    case 'S':
      saltFile = argv[++myarg];
      break;
    case 'l':
      if (1 != sscanf(argv[++myarg], "%zu", &cipherLength))
      {
        fprintf(stderr, "Couldn't parse the length of the cipher text (-l)");
        return 1;
      }
      break;
    case 'L':
      if (1 != sscanf(argv[++myarg], "%zu", &saltLength))
      {
        fprintf(stderr, "Couldn't parse the length of the salt (-L)");
        return 1;
      }
      break;
    case 'T':
      knownFile = argv[++myarg];
      break;
    case 'I':
      if (1 != sscanf(argv[++myarg], "%zu", &knownLength))
      {
        fprintf(stderr, "Couldn't parse the length of known text (-T)");
        return 1;
      }
      break;
    default:
      fprintf(stderr, "Unrecognized command line argument");
      return 1;
    }
  }
  if ((knownPlainText == 0 && knownFile == 0) || (fileName == 0 && (plainText == 0 || knownKey == 0)))
  {
    fprintf(stderr, "You must specify the -t (the known plaintext) or -T (known text in a file) and either -F (a file name to decrypt) or -p (plain text to encrypt) and -k (a key to encrypt your plaintext)");
    return 1;
  }
  if (fileName != 0 && (plainText != 0 || knownKey != 0))
  {
    fprintf(stderr, "Specify either -F, for an input file, or -k and -p for a key and plaintext");
    return 1;
  }
  if (saltFile != 0 && noNullSalt != 0)
  {
    fprintf(stderr, "Specify either -S, for a salt from a file, or -s for a salt from the commandline");
  }
  if (knownFile != 0 && knownPlainText != 0)
  {
    fprintf(stderr, "Specify either -T, for a known text from a file, or -t for known text from the commandline");
  }
  // Get the salt if we have one
  if (saltFile != 0 || noNullSalt != 0) // do we have a salt specified?
  {
    if (saltFile != 0)
    {
      if (saltLength == 0)
      {
        fprintf(stderr, "Salt from file length not specified (-L) defaulting to 11 bytes");
        saltLength = 11;
      }
      FILE *saltPtr;
      saltPtr = fopen(saltFile, "rb");
      if (saltPtr == 0)
      {
        fprintf(stderr, "Could open file %s", saltFile);
        return 1;
      }
      actualSalt = (unsigned char *)malloc(sizeof(unsigned char) * saltLength);
      if (saltLength != fread(actualSalt, sizeof(unsigned char), saltLength, saltPtr))
      {
        fprintf(stderr, "Could not read all of %s", saltFile);
        return 1;
      }
      fclose(saltPtr);
    }
    else
    {
      saltLength = strlen(noNullSalt);
      actualSalt = (unsigned char *)malloc(sizeof(unsigned char) * saltLength);
      actualSalt = (unsigned char *)noNullSalt;
    }
  }
  // Get the cipher text
  if (fileName != 0)
  {
    if (cipherLength == 0)
    {
      fprintf(stderr, "Ciphertext length not specified (-l) defaulting to 128 bytes");
      cipherLength = 128;
    }
    FILE *cipherPtr;
    cipherPtr = fopen(fileName, "rb");
    if (cipherPtr == 0)
    {
      fprintf(stderr, "Could open file %s", fileName);
      return 1;
    }
    cipherText = (unsigned char *)malloc(sizeof(unsigned char) * cipherLength);
    if (cipherLength != fread(cipherText, sizeof(unsigned char), cipherLength, cipherPtr))
    {
      fprintf(stderr, "Could not read all of %s", fileName);
      return 1;
    }
    fclose(cipherPtr);
  }
  else
  {
    cipherLength = strlen(plainText);
    cipherText = (unsigned char *)malloc(cipherLength);
    cipherText = (unsigned char *)plainText;
    keyLength = strlen(knownKey);
    if (saltLength + keyLength > host_max_key)
    {
      fprintf(stderr, "The length of the specified key (or key + salt if specified) is greater than the max key length");
      return 1;
    }
    if (saltLength != 0) // if we have salt, append it
    {
      unsigned char *tempKey = (unsigned char *)malloc(sizeof(unsigned char) * (saltLength + keyLength));
      memcpy(tempKey, knownKey, keyLength);
      memcpy(tempKey + keyLength, actualSalt, saltLength);
      keyLength = keyLength + saltLength;
      prepare_key(tempKey, keyLength, s_box);
    }
    else
    {
      prepare_key((unsigned char *)knownKey, keyLength, s_box); // we know there are no nulls in the key from the command line
    }
    rc4(cipherText, cipherLength, s_box);
  }
  // Get the known plain text
  if (knownFile != 0)
  {
    if (knownLength == 0)
    {
      fprintf(stderr, "Known plain text from file length not specified (-I) defaulting to 8 bytes");
      knownLength = 8;
    }
    FILE *knownPtr;
    knownPtr = fopen(knownFile, "rb");
    if (knownPtr == 0)
    {
      fprintf(stderr, "Could open file %s", knownFile);
      return 1;
    }
    actualPlainText = (unsigned char *)malloc(sizeof(unsigned char) * knownLength);
    if (knownLength != fread(actualPlainText, sizeof(unsigned char), knownLength, knownPtr))
    {
      fprintf(stderr, "Could not read all of %s", knownFile);
      return 1;
    }
    fclose(knownPtr);
    knownPlainText = (char *)actualPlainText;
  }
  else
  {
    knownLength = strlen(knownPlainText);
    actualPlainText = (unsigned char *)malloc(sizeof(unsigned char) * knownLength);
    actualPlainText = (unsigned char *)knownPlainText;
  }
  if (knownLength >= host_max_known)
  {
    fprintf(stderr, "Known plain text is longer than MAX_KNOWN_STREAM_LEN");
    return 1;
  }

  unsigned char *knownKeyStream = (unsigned char *)malloc(sizeof(unsigned char) * knownLength);
  for (int i = 0; i < knownLength; i++)
  {
    knownKeyStream[i] = knownPlainText[i] ^ cipherText[i];
    fprintf(stderr, "- %02X\n", knownKeyStream[i]);
  }
  for (int i = 0; i < saltLength; i++) {
      fprintf(stderr, "salt[%i] = %02x\n", i, actualSalt[i]);
  }

  unsigned char *key = (unsigned char *)malloc(sizeof(unsigned char) * (MAX_KEY_LENGTH + 1));

  hipEvent_t start, stop;
  hipError_t cudaStatus = hipEventCreate(&start);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipEventCreate(start) failed!");
    return 1;
  }
  cudaStatus = hipEventCreate(&stop);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipEventCreate(stop) failed!");
    return 1;
  }

  cudaStatus = hipEventRecord(start, 0);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipEventRecord(start) failed!");
    return 1;
  }

  size_t foundKeyLen = 0;
  // Since null can be the value of any byte of the key we have to know the length
  cudaStatus = crackRc4WithCuda(knownKeyStream, knownLength, key, &foundKeyLen, actualSalt, saltLength);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "addWithCuda failed!");
    return 1;
  }

  cudaStatus = hipEventRecord(stop, 0);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipEventRecord(stop) failed!");
    return 1;
  }

  cudaStatus = hipEventSynchronize(stop);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipEventSynchronize failed!");
    return 1;
  }
  float useTime;
  cudaStatus = hipEventElapsedTime(&useTime, start, stop);
  useTime /= 1000;
  printf("The time we used was:%fs\n", useTime);
  if (foundKeyLen != 0)
  {
    printf("The right key has been found.The right key is:\n");
    for (int idx = 0; idx < foundKeyLen; idx++) {
        printf("%02X", key[idx]);
    }
    printf("\n");
#if 0
    prepare_key(key, foundKeyLen, s_box);

    rc4(cipherText, cipherLength, s_box);

    std::ofstream outf("decrypted");
    outf.write((char *)cipherText, cipherLength);
    outf.close();
    std::ofstream outk("outkey");
    outk.write((char *)key, foundKeyLen);
    printf("\nThe clear text is:\n%s\n", cipherText);
#endif
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Free all of our input variables
  free(key);
  free(knownKeyStream);
  free(s_box);

  hipDeviceReset();
  return 0;
}
